#include "neural_layer.h"


neural_layer *gpu_new_neural_layer(int input_length,
                                   int num_weights,
                                   int num_inputs) {

    neural_layer *layer;

    layer = (neural_layer *) malloc(sizeof(neural_layer));

    layer->w = gpu_new_matrix(num_weights, input_length);

    layer->w_T = gpu_new_matrix(input_length, num_weights);

    layer->s = gpu_new_matrix(num_weights, num_inputs);

    layer->output = gpu_new_matrix(num_weights, num_inputs);

    layer->dL_ds = gpu_new_matrix(num_weights, num_inputs);

    layer->dL_dw = gpu_new_matrix(num_weights, input_length);

    return layer;

}



void gpu_free_neural_layer(neural_layer *layer) {

    hipFree(layer->w);
    hipFree(layer->w_T);
    hipFree(layer->s);
    hipFree(layer->output);
    hipFree(layer->dL_ds);
    hipFree(layer->dL_dw);

    free(layer);

}
